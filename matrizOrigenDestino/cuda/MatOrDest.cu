
#include <hip/hip_runtime.h>
#include<iostream>
#include<string>
using namespace std;

int tc = 0;
int ts = 0;

__host__ int* MakeCarretera(){
    //srand(time(NULL));
    int *res  = (int *)malloc((tc + 1) * sizeof(int));
    for(int i = 0; i < tc ; i++){

        res[i] = (rand() % 10) % 2 ;
    }
    return res;
}

__host__ void MostrarCarretera(int* crt){

    for(int i = 0; i < tc; i++){

        cout << crt[i] << " ";
    }
    cout << "\n";

}

__global__ void prueba(int *ctr,int n){

  //cout << to_string(blockIdx.x) + "\n";
    ctr[blockIdx.x] +=2;
    /*
    int index = threadIdx.x +  blockIdx.x * blockDim.x;
    if (index < n)
      ctr[index] +=1;*/
    //threadIdx.x
}

int main(int argc, char *argv[]) {
  int *carretera, *res;
  int *d_carretera;
  int size = tc + 1;
  int sizeint = size * sizeof(int);
  try{

      tc = stoi(argv[1]);// el tamaño de la carretera
      ts = stoi(argv[2]);// el numero de iteraciones
  }
  catch(const exception& e)
  {

      cout << "No se ingresaron variables, no se hara nada \n\t tc : tamaño de la carreter entero \n\t ts numero de iteraciones que va realizar la simulacion entero"<< '\n';
  }


  if (tc != 0 && ts != 0)  {
    carretera = MakeCarretera();
    MostrarCarretera(carretera);

    //allocate space for devide copies of carretera
    hipMalloc((void **) &d_carretera, sizeint);


    //copiando datos al device
    hipMemcpy(d_carretera, carretera, sizeint ,hipMemcpyHostToDevice);
    //int hilos = 10;
    // lanzar una funcion
    //prueba<<<tc/hilos, hilos >>>(carretera, tc);
    prueba<<<1, tc >>>(carretera, tc);
    //copiando resultados desde el device al host
    hipMemcpy(carretera, d_carretera, sizeint, hipMemcpyDeviceToHost);
    MostrarCarretera(d_carretera);
    //liberando el espacio
    free(carretera);
    hipFree(d_carretera);

  }

  return 0;

}
